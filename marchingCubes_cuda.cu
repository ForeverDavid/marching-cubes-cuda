#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
// MARCHING CUBES															 //
///////////////////////////////////////////////////////////////////////////////
// CS179 - SPRING 2014
// Final project
// Victor Ceballos Inza

// This file contains the CUDA kernels used in the algorithm, as well as the
// functions that initialize and clean up the VBOs.

///////////////////////////////////////////////////////////////////////////////
// Includes																	 //
///////////////////////////////////////////////////////////////////////////////
#include "marchingCubes_cuda.cuh"
#include <hip/hip_vector_types.h>
#include <stdio.h>

///////////////////////////////////////////////////////////////////////////////
// Declarations																 //
///////////////////////////////////////////////////////////////////////////////
#define BLOCK_SIZE 512
#define PI 3.141592654f

#define gpuErrchk(ans) { gpuAssert((ans), (char*)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line,
		bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
				line);
		if (abort)
			exit(code);
	}
}

// Rendering variables
float xmax = 10.0f;
float xmin = -10.0f;
int numPoints = 3;
int dim = 2;
int func = 0;

// Flag to toggle CUDA usage
int cuda = 1;


///////////////////////////////////////////////////////////////////////////////
// Marching cubes table data												 //
///////////////////////////////////////////////////////////////////////////////

__device__
int pointTable[4][1] = {
		{-1}, {0}, {0}, {-1},
};

__device__
int lineTable[16][4] = {
		{-1, -1, -1, -1},
		{ 0,  3, -1, -1},
		{ 0,  1, -1, -1},
		{ 3,  1, -1, -1},
		{ 1,  2, -1, -1},
		{ 0,  1,  3,  2},
		{ 0,  2, -1, -1},
		{ 3,  2, -1, -1},
		{ 3,  2, -1, -1},
		{ 0,  2, -1, -1},
		{ 0,  3,  1,  2},
		{ 1,  2, -1, -1},
		{ 3,  1, -1, -1},
		{ 0,  1, -1, -1},
		{ 0,  3, -1, -1},
		{-1, -1, -1, -1},
};

__device__
int triangleTable[256][15] = {
	{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1},
	{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1},
	{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1},
	{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1},
	{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1},
	{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1},
	{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1},
	{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1},
	{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1},
	{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1},
	{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1},
	{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1},
	{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1},
	{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1},
	{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1},
	{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1},
	{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1},
	{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1},
	{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1},
	{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1},
	{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1},
	{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1},
	{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1},
	{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1},
	{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1},
	{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1},
	{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1},
	{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1},
	{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1},
	{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1},
	{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1},
	{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1},
	{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1},
	{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1},
	{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1},
	{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0},
	{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0},
	{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1},
	{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1},
	{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1},
	{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1},
	{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1},
	{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1},
	{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1},
	{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1},
	{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1},
	{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1},
	{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1},
	{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1},
	{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1},
	{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1},
	{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1},
	{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1},
	{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9},
	{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1},
	{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1},
	{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1},
	{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6},
	{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1},
	{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11},
	{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7},
	{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1},
	{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1},
	{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1},
	{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1},
	{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1},
	{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1},
	{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1},
	{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1},
	{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1},
	{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1},
	{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1},
	{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1},
	{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1},
	{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1},
	{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1},
	{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1},
	{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1},
	{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1},
	{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1},
	{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9},
	{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1},
	{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1},
	{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7},
	{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11},
	{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1},
	{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6},
	{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1},
	{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1},
	{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1},
	{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1},
	{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1},
	{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1},
	{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1},
	{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1},
	{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1},
	{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1},
	{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1},
	{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1},
	{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1},
	{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1},
	{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1},
	{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1},
	{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1},
	{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3},
	{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1},
	{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1},
	{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1},
	{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1},
	{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1},
	{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3},
	{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1},
	{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1},
	{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1},
	{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1},
	{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1},
	{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1},
	{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6},
	{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1},
	{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1},
	{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1},
	{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8},
	{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1},
	{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4},
	{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10},
	{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1},
	{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1},
	{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1},
	{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1},
	{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1},
	{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1},
	{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10},
	{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5},
	{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1},
	{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1},
	{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1},
	{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8},
	{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6},
	{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1},
	{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1},
	{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1},
	{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1},
	{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1},
	{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1},
	{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2},
	{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1},
	{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1},
	{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1},
	{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2},
	{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1},
	{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1},
	{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1},
	{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1},
	{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1},
	{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4},
	{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1},
	{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11},
	{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5},
	{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1},
	{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1},
	{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9},
	{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1},
	{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1},
	{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1},
	{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1},
	{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1},
	{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1},
	{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4},
	{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1},
	{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3},
	{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1},
	{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1},
	{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1},
	{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7},
	{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10},
	{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1},
	{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1},
	{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1},
	{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1},
	{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1},
	{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1},
	{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1},
	{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1},
	{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
	{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
};

__device__
int cube_edgeToVerts[12][2] = {
	{0,1}, {1,2}, {2,3}, {3,0},
	{4,5}, {5,6}, {6,7}, {7,4},
	{0,4}, {1,5}, {2,6}, {3,7},
};

///////////////////////////////////////////////////////////////////////////////
// Surface functions														 //
///////////////////////////////////////////////////////////////////////////////

// Surface to be rendered in 1D
__device__ __host__
int function1D(float4& point, int func)
{
	float fun; int flag;
	switch (func) {

	case 0:
		fun = point.x * point.x;
		flag = (fun < 9);
		break;

	case 1:
		fun = point.x;
		flag = (fun < 0) and (fun > -10);
		break;

	case 2:
		fun = point.x;
		flag = ( (fun > -9) and (fun < -8) ) or ( (fun > 3) and (fun < 7) );
		break;
	}

	return flag;
}

// Surface to be rendered in 2D
__device__ __host__
int function2D(float4& point, int func)
{
	float fun; int flag;
	float c1,c2,c3,c4,c5;
	switch (func) {

	case 0:
		fun = point.x * point.x + point.y * point.y;
		flag = (fun < 9);
		break;

	case 1:
		c1 = (point.x+9)*(point.x+9) + (point.y-9)*(point.y-9);
		c2 = (point.x-0)*(point.x-0) + (point.y-0)*(point.y-0);
		c3 = (point.x+4)*(point.x+4) + (point.y-6)*(point.y-6);
		c4 = (point.x-5)*(point.x-5) + (point.y-5)*(point.y-5);
		c5 = (point.x+7)*(point.x+7) + (point.y+13)*(point.y+13);
		flag = (c1 < 1) or (c2 < 2.25) or (c3 < 4) or (c4 < 25)
			or (c5 < 64 and -10<point.x and -10<point.y)
			or (2<point.x and point.x<4 and -5<point.y and point.y<-3)
			or (point.y < point.x - 15.0 and point.x<10 and -10<point.y);
		break;

	case 2:
		fun = log((double)point.x);
		flag = abs(point.y) < abs(fun) and point.x > 0 and point.x < 10;
		break;
	}

	return flag;
}

// Surface to be rendered in 3D
__device__ __host__
int function3D(float4& point, int func)
{
	float fun; int flag;
	switch (func) {

	case 0:
		fun = point.x * point.x + point.y * point.y + point.z * point.z;
		flag = (fun < 9);
		break;

	case 1:
		fun = point.x * point.x / 5.0 + point.y * point.y / 3.0
			- point.z * point.z / 7.0;
		flag = (fun < 5);
		break;

	case 2:
		fun = point.x * point.x / 10.0 - point.y * point.y / 3.0
			- point.z / 2.0;
		flag = (fun < 0);
		break;
	}

	return flag;
}


///////////////////////////////////////////////////////////////////////////////
// CUDA kernels																 //
///////////////////////////////////////////////////////////////////////////////

// This kernel checks whether each point lies within the desired surface.
__global__
void points_kernel(float4* points, int size, int dim, int func)
{
	// Get unique thread id
	unsigned int globalID = blockIdx.x * blockDim.x + threadIdx.x;

	// Check whether the point lies within.
	// Fourth coordinate represent containment.
	switch (dim) {

	case 1:

		for (int k = globalID; k < size; k += gridDim.x * blockDim.x) {
			float4 pt = points[k];
			points[k].w = function1D(pt, func);
		}
		break;

	case 2:

		for (int k = globalID; k < size * size; k += gridDim.x * blockDim.x) {
			float4 pt = points[k];
			points[k].w = function2D(pt, func);
		}
		break;

	case 3:

		for (int k = globalID; k < size * size * size;
				k += gridDim.x * blockDim.x) {
			float4 pt = points[k];
			points[k].w = function3D(pt, func);
		}
		break;
	}
}

// This kernel classifies each interval in the grid.
__global__
void kernel1D(float4* points, float4* geom, int size)
{
	// Get unique thread ID, this is the point ID
	unsigned int globalID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int id = globalID; id < size; id += gridDim.x * blockDim.x) {

		// Point ID equals interval ID
		if (id < (size - 1) ) {

			// Get the vertices of this interval
			float4 verts[2];
			verts[0] = points[id];
			verts[1] = points[id + 1];

			// Obtain the type of this interval
			int type = 0;
			for (int l = 0; l < 2; l++) {
				type += verts[l].w * pow((double)2,(double)l);
			}

			// Get the configuration for this type of interval from the table
			// and generate the points accordingly
			int* config = pointTable[type];
			int e = config[0];
			if (e != -1) {
				geom[id] = ( verts[e%2] + verts[(e+1)%2] ) * (0.5f);
				geom[id].w = 1.0f;
			}

		}
	}
}

// This kernel classifies each square in the grid.
__global__
void kernel2D(float4* points, float4* geom, int size)
{
	// Get unique thread ID, this is the point ID
	unsigned int globalID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int id = globalID; id < size * size; id += gridDim.x * blockDim.x) {

		// Transform point ID to square ID
		int j = (int) floor((double) (id / size));
		int idx = id - j;

		if (idx < (size - 1) * (size - 1)) {

			// Get the vertices of this square
			float4 verts[4];
			verts[0] = points[id];
			verts[1] = points[id + 1];
			verts[2] = points[id + size + 1];
			verts[3] = points[id + size];

			// Obtain the type of this square
			int type = 0;
			for (int l = 0; l < 4; l++) {
				type += verts[l].w * pow((double)2,(double)l);
			}

			// Get the configuration for this type of square from the table
			// and generate the lines accordingly
			int* config = lineTable[type];
			int e;
			for (int l = 0; l < 4; l++) {
				e = config[l];
				if (e != -1) {
					geom[4*idx + l] = ( verts[e%4] + verts[(e+1)%4] ) * (0.5f);
					geom[4*idx + l].w = 1.0f;
				} else { break; }
			}

		}
	}
}

// This kernel classifies each cube in the grid.
__global__
void kernel3D(float4* points, float4* geom, int size)
{
	// Get unique thread ID, this is the point ID
	unsigned int globalID = blockIdx.x * blockDim.x + threadIdx.x;

	for (int id = globalID; id < size * size * size; id += gridDim.x * blockDim.x) {

		// Transform point ID to cube ID
		int j = (int) ( (int) floor((double) (id / size)) % size );
		int k = (int) floor((double) (id / (size*size)));
		int idx = id - j + k - 2 * k * size;

		if (idx < (size - 1) * (size - 1) * (size - 1)) {

			// Get the vertices of this cube
			float4 verts[8];
			verts[0] = points[id];
			verts[1] = points[id + 1];
			verts[2] = points[id + size + 1];
			verts[3] = points[id + size];

			verts[4] = points[id + size*size];
			verts[5] = points[id + size*size + 1];
			verts[6] = points[id + size*size + size + 1];
			verts[7] = points[id + size*size + size];

			// Obtain the type of this cube
			int type = 0;
			for (int l = 0; l < 8; l++) {
				type += verts[l].w * pow((double)2,(double)l);
			}

			// Get the configuration for this type of cube from the table
			// and generate the triangles accordingly
			int* config = triangleTable[type];
			int e, e0, e1;
			for (int l = 0; l < 15; l++) {
				e = config[l];
				e0 = cube_edgeToVerts[e][0]; e1 = cube_edgeToVerts[e][1];
				if (e != -1) {
					geom[15*idx + l] = ( verts[e0] + verts[e1] ) * (0.5f);
					geom[15*idx + l].w = 1.0f;
				} else { break; }
			}

		}
	}
}


///////////////////////////////////////////////////////////////////////////////
// Run the CUDA part of the computation										 //
///////////////////////////////////////////////////////////////////////////////
void runCuda(GLuint *vbo)
{
	// Map OpenGL buffer object for writing from CUDA
	float4* dev_points;
	float4* dev_geometry;

	// Map OpenGL buffers to CUDA
	cudaGLMapBufferObject((void**) &dev_points, vbo[1]);
	cudaGLMapBufferObject((void**) &dev_geometry, vbo[2]);

	// Choose a block size and a grid size
	const unsigned int threadsPerBlock = BLOCK_SIZE;
	const unsigned int maxBlocks = 50;
	unsigned int blocks;

	// Execute CUDA kernels
	switch (dim) {

	case 1:

		blocks = min(maxBlocks,
				(int) ceil(numPoints / (float) threadsPerBlock));

		// Check for containment of vertices
		points_kernel<<<blocks, threadsPerBlock>>>
				(dev_points, numPoints, dim, func);

		// Obtain the edges from the data table
		kernel1D<<<blocks, threadsPerBlock>>>
				(dev_points, dev_geometry, numPoints);

		break;

	case 2:

		blocks = min(maxBlocks,
				(int) ceil(numPoints * numPoints / (float) threadsPerBlock));

		// Check for containment of vertices
		points_kernel<<<blocks, threadsPerBlock>>>
				(dev_points, numPoints, dim, func);

		// Obtain the edges from the data table
		kernel2D<<<blocks, threadsPerBlock>>>
				(dev_points, dev_geometry, numPoints);

		break;

	case 3:

		blocks = min(maxBlocks,
				(int) ceil(
						numPoints * numPoints * numPoints
								/ (float) threadsPerBlock));

		// Check for containment of vertices
		points_kernel<<<blocks, threadsPerBlock>>>
				(dev_points, numPoints, dim, func);

		// Obtain the triangles from the data table
		kernel3D<<<blocks, threadsPerBlock>>>
				(dev_points, dev_geometry, numPoints);

		break;
	}

	// Unmap buffer objects from CUDA
	cudaGLUnmapBufferObject(vbo[1]);
	cudaGLUnmapBufferObject(vbo[2]);
}


///////////////////////////////////////////////////////////////////////////////
// Vertex Buffer Objects													 //
///////////////////////////////////////////////////////////////////////////////

// Initialize 1D data
void createData1D(float4* points, float4* grid, float4* geom)
{
	// Initialize points data.
	float delta = (xmax - xmin) / (numPoints - 1);
	for (int i = 0; i < numPoints; i++) {

		// Set initial position data
		points[i].x = xmin + delta * i;
		points[i].y = 0.0f;
		points[i].z = 0.0f;
		points[i].w = 1.0f;
	}

	// Initialize grid data.
	for (int i = 0; i < (numPoints - 1); i++) {

		// Set initial position data
		grid[2*i+0] = points[i];
		grid[2*i+1] = points[i+1];
	}

	// Initialize geometry data.
	float4 zero = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	for (int k = 0; k < (numPoints - 1); k++) {
		geom[k] = zero;
	}
}

// Initialize 2D data
void createData2D(float4* points, float4* grid, float4* geom)
{
	// Initialize points data.
	float delta = (xmax - xmin) / (numPoints - 1);
	for (int i = 0; i < numPoints; i++) {
		for (int j = 0; j < numPoints; j++) {

			int idx = i + j * numPoints;

			// Set initial position data
			points[idx].x = xmin + delta * i;
			points[idx].y = xmax - delta * j;
			points[idx].z = 0.0f;
			points[idx].w = 1.0f;
		}
	}

	// Initialize grid data.
	for (int i = 0; i < (numPoints - 1); i++) {
		for (int j = 0; j < (numPoints - 1); j++) {

			int idx_pt = i + j * numPoints;
			int idx_sq = idx_pt - j;

			// Set initial position data
			grid[4 * idx_sq + 0] = points[idx_pt];
			grid[4 * idx_sq + 1] = points[idx_pt + 1];
			grid[4 * idx_sq + 2] = points[idx_pt + numPoints + 1];
			grid[4 * idx_sq + 3] = points[idx_pt + numPoints];
		}
	}

	// Initialize geometry data.
	float4 zero = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	for (int k = 0; k < (numPoints - 1) * (numPoints - 1) * 4; k++) {
		geom[k] = zero;
	}
}

// Initialize 3D data
void createData3D(float4* points, float4* grid, float4* geom)
{
	// Initialize points data.
	float delta = (xmax - xmin) / (numPoints - 1);
	for (int i = 0; i < numPoints; i++) {
		for (int j = 0; j < numPoints; j++) {
			for (int k = 0; k < numPoints; k++) {

				int idx = i + j * numPoints + k * numPoints * numPoints;

				// Set initial position data
				points[idx].x = xmin + delta * i;
				points[idx].y = xmax - delta * j;
				points[idx].z = xmin + delta * k;
				points[idx].w = 1.0f;
			}
		}
	}

	// Initialize grid data.
	for (int i = 0; i < (numPoints - 1); i++) {
		for (int j = 0; j < (numPoints - 1); j++) {
			for (int k = 0; k < (numPoints - 1); k++) {

				int idx_pt = i + j * numPoints + k * numPoints * numPoints;
				int idx_sq = idx_pt - j + k - 2 * k * numPoints;

				// Set initial position data
				grid[16 * idx_sq + 0] = points[idx_pt];
				grid[16 * idx_sq + 1] = points[idx_pt+1];
				grid[16 * idx_sq + 2] = points[idx_pt+numPoints+1];
				grid[16 * idx_sq + 3] = points[idx_pt+numPoints];

				grid[16 * idx_sq + 4] = points[idx_pt+numPoints*numPoints];
				grid[16 * idx_sq + 5] = points[idx_pt+numPoints*numPoints+1];
				grid[16 * idx_sq + 6] = points[idx_pt+numPoints*numPoints+numPoints+1];
				grid[16 * idx_sq + 7] = points[idx_pt+numPoints*numPoints+numPoints];

				grid[16 * idx_sq + 8] = points[idx_pt];
				grid[16 * idx_sq + 9] = points[idx_pt+1];
				grid[16 * idx_sq + 10] = points[idx_pt+numPoints*numPoints+1];
				grid[16 * idx_sq + 11] = points[idx_pt+numPoints*numPoints];

				grid[16 * idx_sq + 12] = points[idx_pt+numPoints];
				grid[16 * idx_sq + 13] = points[idx_pt+numPoints+1];
				grid[16 * idx_sq + 14] = points[idx_pt+numPoints*numPoints+numPoints+1];
				grid[16 * idx_sq + 15] = points[idx_pt+numPoints*numPoints+numPoints];

			}
		}
	}

	// Initialize geometry data.
	float4 zero = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	for (int k = 0; k < (numPoints - 1) * (numPoints - 1) * (numPoints - 1) * 15; k++) {
		geom[k] = zero;
	}
}

// Create VBOs
void createVBOs(GLuint* vbo)
{
	// Create VBOs.
	glGenBuffers(3, vbo);

	// Initialize points and grid
	unsigned int points_size;
	float4* points;
	unsigned int grid_size;
	float4* grid;
	unsigned int geom_size;
	float4* geom;

	switch (dim) {

	case 1:

		// Allocate memory
		points_size = numPoints * sizeof(float4);
		points = (float4*) malloc(points_size);
		grid_size = (numPoints - 1) * 2 * sizeof(float4);
		grid = (float4*) malloc(grid_size);
		geom_size = (numPoints - 1) * sizeof(float4);
		geom = (float4*) malloc(geom_size);
		// Initialize data
		createData1D(points, grid, geom);
		break;

	case 2:

		// Allocate memory
		points_size = numPoints * numPoints * sizeof(float4);
		points = (float4*) malloc(points_size);
		grid_size = (numPoints - 1) * (numPoints - 1) * 4 * sizeof(float4);
		grid = (float4*) malloc(grid_size);
		geom_size = (numPoints - 1) * (numPoints - 1) * 4 * sizeof(float4);
		geom = (float4*) malloc(geom_size);
		// Initialize data
		createData2D(points, grid, geom);
		break;

	case 3:

		// Allocate memory
		points_size = numPoints * numPoints * numPoints * sizeof(float4);
		points = (float4*) malloc(points_size);
		grid_size = (numPoints - 1) * (numPoints - 1) * (numPoints - 1) * 16
				* sizeof(float4);
		grid = (float4*) malloc(grid_size);
		geom_size = (numPoints - 1) * (numPoints - 1) * (numPoints - 1) * 15
				* sizeof(float4);
		geom = (float4*) malloc(geom_size);
		// Initialize data
		createData3D(points, grid, geom);
		break;
	}

	// Activate VBO id to use.
	glBindBuffer(GL_ARRAY_BUFFER, vbo[0]);

	// Upload data to video card.
	glBufferData(GL_ARRAY_BUFFER, grid_size, grid, GL_DYNAMIC_DRAW);

	// Activate VBO id to use.
	glBindBuffer(GL_ARRAY_BUFFER, vbo[1]);

	// Upload data to video card.
	glBufferData(GL_ARRAY_BUFFER, points_size, points, GL_DYNAMIC_DRAW);

	// Register buffer objects with CUDA
	gpuErrchk(cudaGLRegisterBufferObject(vbo[1]));

	// Activate VBO id to use.
	glBindBuffer(GL_ARRAY_BUFFER, vbo[2]);

	// Upload data to video card.
	glBufferData(GL_ARRAY_BUFFER, geom_size, geom, GL_DYNAMIC_DRAW);

	// Register buffer objects with CUDA
	gpuErrchk(cudaGLRegisterBufferObject(vbo[2]));

	// Free temporary data
	free(points); free(grid); free(geom);

	// Release VBOs with ID 0 after use.
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// Execute the algorithm, if asked
	if (cuda) { runCuda(vbo); }

}

// Delete VBOs
void deleteVBOs(GLuint* vbo)
{
	// Delete VBOs
	glBindBuffer(1, vbo[0]);
	glDeleteBuffers(1, &vbo[0]);
	glBindBuffer(1, vbo[1]);
	glDeleteBuffers(1, &vbo[1]);
	glBindBuffer(1, vbo[2]);
	glDeleteBuffers(1, &vbo[2]);

	// Unregister buffer objects with CUDA
	gpuErrchk(cudaGLUnregisterBufferObject(vbo[1]));
	gpuErrchk(cudaGLUnregisterBufferObject(vbo[2]));

	// Free VBOs
	*vbo = 0;
}


///////////////////////////////////////////////////////////////////////////////
// Gets/sets the number of vertices											 //
///////////////////////////////////////////////////////////////////////////////
int getNumPoints()
{
	return numPoints;
}
void setNumPoints(int n)
{
	numPoints = n;
}

///////////////////////////////////////////////////////////////////////////////
// Gets/sets the dimension													 //
///////////////////////////////////////////////////////////////////////////////
int getDimension()
{
	return dim;
}
void setDimension(int n)
{
	dim = n;
}

///////////////////////////////////////////////////////////////////////////////
// Sets the GPU usage														 //
///////////////////////////////////////////////////////////////////////////////
void setCUDA()
{
	cuda = 1 - cuda;
}

///////////////////////////////////////////////////////////////////////////////
// Changes the function of the surface to render							 //
///////////////////////////////////////////////////////////////////////////////
void changeFunction()
{
	func = (func+1)%3;
}


